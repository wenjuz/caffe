
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/gpu_memory.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

// Those defines serve single purpose to keep sane C++ formatting
// in presence of <80 characters rule
#define cudnnConvFwd                       cudnnConvolutionForward
#define cudnnConvBwdBias                   cudnnConvolutionBackwardBias
#define cudnnConvBwdFilter                 cudnnConvolutionBackwardFilter_v3
#define cudnnConvBwdData                   cudnnConvolutionBackwardData_v3

namespace caffe {

  __global__ void sync_conv_groups() { }

template <typename Dtype, typename Mtype>
void CuDNNConvolutionLayer<Dtype,Mtype>::Forward_gpu(
    const vector<BlobBase*>& bottom, const vector<BlobBase*>& top) {
    const Dtype* weight = this->blobs_[0]->template gpu_data<Dtype>();
    for (int i = 0; i < bottom.size(); ++i) {
      const Dtype* bottom_data = bottom[i]->gpu_data<Dtype>();
      Dtype* top_data = top[i]->mutable_gpu_data<Dtype>();

      // Test free space and force reshape if allocations have changed
      size_t workspace_limit_bytes, total_memory;
      gpu_memory::getInfo(&workspace_limit_bytes, &total_memory);
      if (workspace_fwd_sizes_[i] > workspace_limit_bytes) {
          this->Reshape(bottom, top);
      }

      // !!!! Not safe if group_ > 1 !!!!
      workspace.reserve(workspace_fwd_sizes_[i]);

      // Forward through cuDNN in parallel over groups.
      for (int g = 0; g < this->group_; g++) {
          // Filters.
        CUDNN_CHECK(cudnnConvFwd(Caffe::cudnn_handle(),
                                 cudnn::dataType<Dtype>::one,
                                 bottom_descs_[i],
                                 bottom_data + bottom_offset_ * g,
                                 filter_desc_,
                                 weight + this->weight_offset_ * g,
                                 conv_descs_[i],
                                 fwd_algo_[i],
                                 workspace.data(),
                                 workspace.size(),
                                 cudnn::dataType<Dtype>::zero,
                                 top_descs_[i],
                                 top_data + top_offset_ * g));

        // Bias.
        if (this->bias_term_) {
          const Dtype* bias_data = this->blobs_[1]->template gpu_data<Dtype>();
          CUDNN_CHECK(cudnnAddTensor_v3(Caffe::cudnn_handle(),
                                        cudnn::dataType<Dtype>::one,
                                        bias_desc_,
                                        bias_data + bias_offset_ * g,
                                        cudnn::dataType<Dtype>::one,
                                        top_descs_[i],
                                        top_data + top_offset_ * g));
        }

      }

      workspace.release();
      // Synchronize the work across groups, each of which went into its own
      // stream, by launching an empty kernel into the default (null) stream.
      // NOLINT_NEXT_LINE(whitespace/operators)
      CUDA_CHECK(cudaStreamSynchronize(cudaStreamLegacy));
    }
  }


template <typename Dtype, typename Mtype>
void CuDNNConvolutionLayer<Dtype,Mtype>::Backward_gpu(const vector<BlobBase*>& top,
    const vector<bool>& propagate_down, const vector<BlobBase*>& bottom) {
    const Dtype* weight = NULL;
    Dtype* weight_diff = NULL;


    if (this->param_propagate_down_[0]) {
      weight = this->blobs_[0]->template gpu_data<Dtype>();
      weight_diff = this->blobs_[0]->template mutable_gpu_diff<Dtype>();
    }
    Dtype* bias_diff = NULL;

    if (this->bias_term_ && this->param_propagate_down_[1]) {
      bias_diff = this->blobs_[1]->template mutable_gpu_diff<Dtype>();
    }

    for (int i = 0; i < top.size(); ++i) {
      const Dtype* top_diff = top[i]->gpu_diff<Dtype>();

        // Test free space and force reshape if allocations have changed
        size_t workspace_limit_bytes, total_memory;
        gpu_memory::getInfo(&workspace_limit_bytes, &total_memory);
        if (workspace_bwd_filter_sizes_[i] > workspace_limit_bytes ||
           workspace_bwd_data_sizes_[i] > workspace_limit_bytes) {
            this->Reshape(bottom, top);
        }

        // To remove pressure on allocator, allocate the larger of the
        // workspaces needed for the following steps
        size_t workspace_reserve = workspace_bwd_filter_sizes_[i] >
            workspace_bwd_data_sizes_[i] ?
            workspace_bwd_filter_sizes_[i] : workspace_bwd_data_sizes_[i];

        // !!!! Not safe if group_ > 1 !!!!
        workspace.reserve(workspace_reserve);

        // Backward through cuDNN in parallel over groups and gradients.
        for (int g = 0; g < this->group_; g++) {
            // Gradient w.r.t. bias.
            if (this->bias_term_ && this->param_propagate_down_[1]) {
                CUDNN_CHECK(cudnnConvBwdBias(Caffe::cudnn_handle(),
                                             cudnn::dataType<Dtype>::one,
                                             top_descs_[i],
                                             top_diff + top_offset_ * g,
                                             cudnn::dataType<Dtype>::one,
                                             bias_desc_,
                                             bias_diff + bias_offset_ * g));
            }

            // Gradient w.r.t. weights.
            if (this->param_propagate_down_[0]) {
          const Dtype* bottom_data = bottom[i]->gpu_data<Dtype>();
                CUDNN_CHECK(cudnnConvBwdFilter(Caffe::cudnn_handle(),
                                          cudnn::dataType<Dtype>::one,
                                          bottom_descs_[i],
                                          bottom_data + bottom_offset_ * g,
                                          top_descs_[i],
                                          top_diff + top_offset_ * g,
                                          conv_descs_[i],
                                          bwd_filter_algo_[i],
                                          workspace.data(),
                                          workspace.size(),
                                          cudnn::dataType<Dtype>::one,
                                          filter_desc_,
                                          weight_diff + weight_offset_ * g));
            }

            // Gradient w.r.t. bottom data.
            if (propagate_down[i]) {
                if (weight == NULL) {
            weight = this->blobs_[0]->template gpu_data<Dtype>();
                }
          Dtype* bottom_diff = bottom[i]->mutable_gpu_diff<Dtype>();
                CUDNN_CHECK(cudnnConvBwdData(Caffe::cudnn_handle(),
                                             cudnn::dataType<Dtype>::one,
                                             filter_desc_,
                                             weight + this->weight_offset_ * g,
                                             top_descs_[i],
                                             top_diff + top_offset_ * g,
                                             conv_descs_[i],
                                             bwd_data_algo_[i],
                                             workspace.data(),
                                             workspace.size(),
                                             cudnn::dataType<Dtype>::zero,
                                             bottom_descs_[i],
                                             bottom_diff + bottom_offset_ * g));
            }
        }

        workspace.release();
        // Synchronize the work across groups, each of which went into its own
        // stream, by launching an empty kernel into the default (null) stream.
        // NOLINT_NEXT_LINE(whitespace/operators)
        CUDA_CHECK(cudaStreamSynchronize(cudaStreamLegacy));
    }
  }

  INSTANTIATE_LAYER_GPU_FUNCS(CuDNNConvolutionLayer);

}  // namespace caffe
#endif
