
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
void CuDNNSoftmaxLayer<Dtype,Mtype>::Forward_gpu(const vector<BlobBase*>& bottom,
    const vector<BlobBase*>& top) {
    SoftmaxLayer<Dtype,Mtype>::Forward_gpu(bottom, top);
}

template <typename Dtype, typename Mtype>
void CuDNNSoftmaxLayer<Dtype,Mtype>::Backward_gpu(const vector<BlobBase*>& top,
    const vector<bool>& propagate_down, const vector<BlobBase*>& bottom) {
    SoftmaxLayer<Dtype,Mtype>::Backward_gpu(top, propagate_down, bottom);
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNSoftmaxLayer);

}  // namespace caffe
#endif
